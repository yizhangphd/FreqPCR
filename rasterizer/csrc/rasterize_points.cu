#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates. All rights reserved.

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <math.h>
#include <cstdio>
#include <sstream>
#include <tuple>
#include "bitmask.cuh"
#include "rasterization_utils.cuh"

namespace {
// A little structure for holding details about a pixel.
struct Pix {
  float z; // Depth of the reference point.
  int32_t idx; // Index of the reference point.
  float dist2; // Euclidean distance square to the reference point.
};

__device__ inline bool operator<(const Pix& a, const Pix& b) {
  return a.z < b.z;
}

// This function checks if a pixel given by xy location pxy lies within the
// point with index p and batch index n. One of the inputs is a list (q)
// which contains Pixel structs with the indices of the points which intersect
// with this pixel sorted by closest z distance. If the pixel pxy lies in the
// point, the list (q) is updated and re-orderered in place. In addition
// the auxillary variables q_size, q_max_z and q_max_idx are also modified.
// This code is shared between RasterizePointsNaiveCudaKernel and
// RasterizePointsFineCudaKernel.
template <typename PointQ>
__device__ void CheckPixelInsidePoint(
    const float* points, // (P, 3)
    const int p_idx,
    int& q_size,
    float& q_max_z,
    int& q_max_idx,
    PointQ& q,
    const float* radius,
    const float xf,
    const float yf,
    const int K) {
  const float px = points[p_idx * 3 + 0];
  const float py = points[p_idx * 3 + 1];
  const float pz = points[p_idx * 3 + 2];
  const float p_radius = radius[p_idx];
  const float radius2 = p_radius * p_radius;
  if (pz < 0)
    return; // Don't render points behind the camera
  const float dx = xf - px;
  const float dy = yf - py;
  const float dist2 = dx * dx + dy * dy;
  if (dist2 < radius2) {
    if (q_size < K) {
      // Just insert it
      q[q_size] = {pz, p_idx, dist2};
      if (pz > q_max_z) {
        q_max_z = pz;
        q_max_idx = q_size;
      }
      q_size++;
    } else if (pz < q_max_z) {
      // Overwrite the old max, and find the new max
      q[q_max_idx] = {pz, p_idx, dist2};
      q_max_z = pz;
      for (int i = 0; i < K; i++) {
        if (q[i].z > q_max_z) {
          q_max_z = q[i].z;
          q_max_idx = i;
        }
      }
    }
  }
}
} // namespace
// ****************************************************************************
// *                          NAIVE RASTERIZATION                             *
// ****************************************************************************

__global__ void RasterizePointsNaiveCudaKernel(
    const float* points, // (P, 3)
    const int64_t* cloud_to_packed_first_idx, // (N)
    const int64_t* num_points_per_cloud, // (N)
    const float* radius,
    const int N,
    const int H,
    const int W,
    const int K,
    int32_t* point_idxs, // (N, H, W, K)
    float* zbuf, // (N, H, W, K)
    float* pix_dists) { // (N, H, W, K)
  // Simple version: One thread per output pixel
  const int num_threads = gridDim.x * blockDim.x;
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;
  for (int i = tid; i < N * H * W; i += num_threads) {
    // Convert linear index to 3D index
    const int n = i / (H * W); // Batch index
    const int pix_idx = i % (H * W);

    // Reverse ordering of the X and Y axis as the camera coordinates
    // assume that +Y is pointing up and +X is pointing left.
    const int yi = H - 1 - pix_idx / W;
    const int xi = W - 1 - pix_idx % W;

    // screen coordinates to ndc coordiantes of pixel.
    const float xf = PixToNonSquareNdc(xi, W, H);
    const float yf = PixToNonSquareNdc(yi, H, W);

    // For keeping track of the K closest points we want a data structure
    // that (1) gives O(1) access to the closest point for easy comparisons,
    // and (2) allows insertion of new elements. In the CPU version we use
    // std::priority_queue; then (2) is O(log K). We can't use STL
    // containers in CUDA; we could roll our own max heap in an array, but
    // that would likely have a lot of warp divergence so we do something
    // simpler instead: keep the elements in an unsorted array, but keep
    // track of the max value and the index of the max value. Then (1) is
    // still O(1) time, while (2) is O(K) with a clean loop. Since K <= 8
    // this should be fast enough for our purposes.
    // TODO(jcjohns) Abstract this out into a standalone data structure
    Pix q[kMaxPointsPerPixel];
    int q_size = 0;
    float q_max_z = -1000;
    int q_max_idx = -1;

    // Using the batch index of the thread get the start and stop
    // indices for the points.
    const int64_t point_start_idx = cloud_to_packed_first_idx[n];
    const int64_t point_stop_idx = point_start_idx + num_points_per_cloud[n];

    for (int p_idx = point_start_idx; p_idx < point_stop_idx; ++p_idx) {
      CheckPixelInsidePoint(
          points, p_idx, q_size, q_max_z, q_max_idx, q, radius, xf, yf, K);
    }
    BubbleSort(q, q_size);
    int idx = n * H * W * K + pix_idx * K;
    for (int k = 0; k < q_size; ++k) {
      point_idxs[idx + k] = q[k].idx;
      zbuf[idx + k] = q[k].z;
      pix_dists[idx + k] = q[k].dist2;
    }
  }
}

std::tuple<at::Tensor, at::Tensor, at::Tensor> RasterizePointsNaiveCuda(
    const at::Tensor& points, // (P. 3)
    const at::Tensor& cloud_to_packed_first_idx, // (N)
    const at::Tensor& num_points_per_cloud, // (N)
    const std::tuple<int, int> image_size,
    const at::Tensor& radius,
    const int points_per_pixel) {
  // Check inputs are on the same device
  at::TensorArg points_t{points, "points", 1},
      cloud_to_packed_first_idx_t{
          cloud_to_packed_first_idx, "cloud_to_packed_first_idx", 2},
      num_points_per_cloud_t{num_points_per_cloud, "num_points_per_cloud", 3};
  at::CheckedFrom c = "RasterizePointsNaiveCuda";
  at::checkAllSameGPU(
      c, {points_t, cloud_to_packed_first_idx_t, num_points_per_cloud_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(points.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  TORCH_CHECK(
      points.ndimension() == 2 && points.size(1) == 3,
      "points must have dimensions (num_points, 3)");
  TORCH_CHECK(
      num_points_per_cloud.size(0) == cloud_to_packed_first_idx.size(0),
      "num_points_per_cloud must have same size first dimension as cloud_to_packed_first_idx");

  const int N = num_points_per_cloud.size(0); // batch size.
  const int H = std::get<0>(image_size);
  const int W = std::get<1>(image_size);
  const int K = points_per_pixel;

  if (K > kMaxPointsPerPixel) {
    std::stringstream ss;
    ss << "Must have points_per_pixel <= " << kMaxPointsPerPixel;
    AT_ERROR(ss.str());
  }

  auto int_opts = num_points_per_cloud.options().dtype(at::kInt);
  auto float_opts = points.options().dtype(at::kFloat);
  at::Tensor point_idxs = at::full({N, H, W, K}, -1, int_opts);
  at::Tensor zbuf = at::full({N, H, W, K}, -1, float_opts);
  at::Tensor pix_dists = at::full({N, H, W, K}, -1, float_opts);

  if (point_idxs.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(point_idxs, zbuf, pix_dists);
  }

  const size_t blocks = 1024;
  const size_t threads = 64;
  RasterizePointsNaiveCudaKernel<<<blocks, threads, 0, stream>>>(
      points.contiguous().data_ptr<float>(),
      cloud_to_packed_first_idx.contiguous().data_ptr<int64_t>(),
      num_points_per_cloud.contiguous().data_ptr<int64_t>(),
      radius.contiguous().data_ptr<float>(),
      N,
      H,
      W,
      K,
      point_idxs.contiguous().data_ptr<int32_t>(),
      zbuf.contiguous().data_ptr<float>(),
      pix_dists.contiguous().data_ptr<float>());

  AT_CUDA_CHECK(hipGetLastError());
  return std::make_tuple(point_idxs, zbuf, pix_dists);
}

// ****************************************************************************
// *                          COARSE RASTERIZATION                            *
// ****************************************************************************

__global__ void RasterizePointsCoarseCudaKernel(
    const float* points, // (P, 3)
    const int64_t* cloud_to_packed_first_idx, // (N)
    const int64_t* num_points_per_cloud, // (N)
    const float* radius,
    const int N,
    const int P,
    const int H,
    const int W,
    const int bin_size,
    const int chunk_size,
    const int max_points_per_bin,
    int* points_per_bin,
    int* bin_points) {
  extern __shared__ char sbuf[];
  const int M = max_points_per_bin;

  // Integer divide round up
  const int num_bins_x = 1 + (W - 1) / bin_size;
  const int num_bins_y = 1 + (H - 1) / bin_size;

  // NDC range depends on the ratio of W/H
  // The shorter side from (H, W) is given an NDC range of 2.0 and
  // the other side is scaled by the ratio of H:W.
  const float NDC_x_half_range = NonSquareNdcRange(W, H) / 2.0f;
  const float NDC_y_half_range = NonSquareNdcRange(H, W) / 2.0f;

  // Size of half a pixel in NDC units is the NDC half range
  // divided by the corresponding image dimension
  const float half_pix_x = NDC_x_half_range / W;
  const float half_pix_y = NDC_y_half_range / H;

  // This is a boolean array of shape (num_bins_y, num_bins_x, chunk_size)
  // stored in shared memory that will track whether each point in the chunk
  // falls into each bin of the image.
  BitMask binmask((unsigned int*)sbuf, num_bins_y, num_bins_x, chunk_size);

  // Have each block handle a chunk of points and build a 3D bitmask in
  // shared memory to mark which points hit which bins.  In this first phase,
  // each thread processes one point at a time. After processing the chunk,
  // one thread is assigned per bin, and the thread counts and writes the
  // points for the bin out to global memory.
  const int chunks_per_batch = 1 + (P - 1) / chunk_size;
  const int num_chunks = N * chunks_per_batch;
  for (int chunk = blockIdx.x; chunk < num_chunks; chunk += gridDim.x) {
    const int batch_idx = chunk / chunks_per_batch;
    const int chunk_idx = chunk % chunks_per_batch;
    const int point_start_idx = chunk_idx * chunk_size;

    binmask.block_clear();

    // Using the batch index of the thread get the start and stop
    // indices for the points.
    const int64_t cloud_point_start_idx = cloud_to_packed_first_idx[batch_idx];
    const int64_t cloud_point_stop_idx =
        cloud_point_start_idx + num_points_per_cloud[batch_idx];

    // Have each thread handle a different point within the chunk
    for (int p = threadIdx.x; p < chunk_size; p += blockDim.x) {
      const int p_idx = point_start_idx + p;

      // Check if point index corresponds to the cloud in the batch given by
      // batch_idx.
      if (p_idx >= cloud_point_stop_idx || p_idx < cloud_point_start_idx) {
        continue;
      }

      const float px = points[p_idx * 3 + 0];
      const float py = points[p_idx * 3 + 1];
      const float pz = points[p_idx * 3 + 2];
      const float p_radius = radius[p_idx];
      if (pz < 0)
        continue; // Don't render points behind the camera.
      const float px0 = px - p_radius;
      const float px1 = px + p_radius;
      const float py0 = py - p_radius;
      const float py1 = py + p_radius;

      // Brute-force search over all bins; TODO something smarter?
      // For example we could compute the exact bin where the point falls,
      // then check neighboring bins. This way we wouldn't have to check
      // all bins (however then we might have more warp divergence?)
      for (int by = 0; by < num_bins_y; ++by) {
        // Get y extent for the bin. PixToNonSquareNdc gives us the location of
        // the center of each pixel, so we need to add/subtract a half
        // pixel to get the true extent of the bin.
        const float by0 = PixToNonSquareNdc(by * bin_size, H, W) - half_pix_y;
        const float by1 =
            PixToNonSquareNdc((by + 1) * bin_size - 1, H, W) + half_pix_y;
        const bool y_overlap = (py0 <= by1) && (by0 <= py1);

        if (!y_overlap) {
          continue;
        }
        for (int bx = 0; bx < num_bins_x; ++bx) {
          // Get x extent for the bin; again we need to adjust the
          // output of PixToNonSquareNdc by half a pixel.
          const float bx0 = PixToNonSquareNdc(bx * bin_size, W, H) - half_pix_x;
          const float bx1 =
              PixToNonSquareNdc((bx + 1) * bin_size - 1, W, H) + half_pix_x;
          const bool x_overlap = (px0 <= bx1) && (bx0 <= px1);

          if (x_overlap) {
            binmask.set(by, bx, p);
          }
        }
      }
    }
    __syncthreads();
    // Now we have processed every point in the current chunk. We need to
    // count the number of points in each bin so we can write the indices
    // out to global memory. We have each thread handle a different bin.
    for (int byx = threadIdx.x; byx < num_bins_y * num_bins_x;
         byx += blockDim.x) {
      const int by = byx / num_bins_x;
      const int bx = byx % num_bins_x;
      const int count = binmask.count(by, bx);
      const int points_per_bin_idx =
          batch_idx * num_bins_y * num_bins_x + by * num_bins_x + bx;

      // This atomically increments the (global) number of points found
      // in the current bin, and gets the previous value of the counter;
      // this effectively allocates space in the bin_points array for the
      // points in the current chunk that fall into this bin.
      const int start = atomicAdd(points_per_bin + points_per_bin_idx, count);

      // Now loop over the binmask and write the active bits for this bin
      // out to bin_points.
      int next_idx = batch_idx * num_bins_y * num_bins_x * M +
          by * num_bins_x * M + bx * M + start;
      for (int p = 0; p < chunk_size; ++p) {
        if (binmask.get(by, bx, p)) {
          // TODO: Throw an error if next_idx >= M -- this means that
          // we got more than max_points_per_bin in this bin
          // TODO: check if atomicAdd is needed in line 265.
          bin_points[next_idx] = point_start_idx + p;
          next_idx++;
        }
      }
    }
    __syncthreads();
  }
}

at::Tensor RasterizePointsCoarseCuda(
    const at::Tensor& points, // (P, 3)
    const at::Tensor& cloud_to_packed_first_idx, // (N)
    const at::Tensor& num_points_per_cloud, // (N)
    const std::tuple<int, int> image_size,
    const at::Tensor& radius,
    const int bin_size,
    const int max_points_per_bin) {
  TORCH_CHECK(
      points.ndimension() == 2 && points.size(1) == 3,
      "points must have dimensions (num_points, 3)");

  // Check inputs are on the same device
  at::TensorArg points_t{points, "points", 1},
      cloud_to_packed_first_idx_t{
          cloud_to_packed_first_idx, "cloud_to_packed_first_idx", 2},
      num_points_per_cloud_t{num_points_per_cloud, "num_points_per_cloud", 3};
  at::CheckedFrom c = "RasterizePointsCoarseCuda";
  at::checkAllSameGPU(
      c, {points_t, cloud_to_packed_first_idx_t, num_points_per_cloud_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(points.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int H = std::get<0>(image_size);
  const int W = std::get<1>(image_size);

  const int P = points.size(0);
  const int N = num_points_per_cloud.size(0);
  const int M = max_points_per_bin;

  // Integer divide round up.
  const int num_bins_y = 1 + (H - 1) / bin_size;
  const int num_bins_x = 1 + (W - 1) / bin_size;

  if (num_bins_y >= kMaxItemsPerBin || num_bins_x >= kMaxItemsPerBin) {
    // Make sure we do not use too much shared memory.
    std::stringstream ss;
    ss << "In Coarse Rasterizer got num_bins_y: " << num_bins_y
       << ", num_bins_x: " << num_bins_x << ", "
       << "; that's too many!";
    AT_ERROR(ss.str());
  }
  auto opts = num_points_per_cloud.options().dtype(at::kInt);
  at::Tensor points_per_bin = at::zeros({N, num_bins_y, num_bins_x}, opts);
  at::Tensor bin_points = at::full({N, num_bins_y, num_bins_x, M}, -1, opts);

  if (bin_points.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return bin_points;
  }

  const int chunk_size = 512;
  const size_t shared_size = num_bins_y * num_bins_x * chunk_size / 8;
  const size_t blocks = 64;
  const size_t threads = 512;

  RasterizePointsCoarseCudaKernel<<<blocks, threads, shared_size, stream>>>(
      points.contiguous().data_ptr<float>(),
      cloud_to_packed_first_idx.contiguous().data_ptr<int64_t>(),
      num_points_per_cloud.contiguous().data_ptr<int64_t>(),
      radius.contiguous().data_ptr<float>(),
      N,
      P,
      H,
      W,
      bin_size,
      chunk_size,
      M,
      points_per_bin.contiguous().data_ptr<int32_t>(),
      bin_points.contiguous().data_ptr<int32_t>());

  AT_CUDA_CHECK(hipGetLastError());
  return bin_points;
}

// ****************************************************************************
// *                            FINE RASTERIZATION                            *
// ****************************************************************************

__global__ void RasterizePointsFineCudaKernel(
    const float* points, // (P, 3)
    const int32_t* bin_points, // (N, BH, BW, T)
    const float* radius,
    const int bin_size,
    const int N,
    const int BH, // num_bins y
    const int BW, // num_bins x
    const int M,
    const int H,
    const int W,
    const int K,
    int32_t* point_idxs, // (N, H, W, K)
    float* zbuf, // (N, H, W, K)
    float* pix_dists) { // (N, H, W, K)
  // This can be more than H * W if H or W are not divisible by bin_size.
  const int num_pixels = N * BH * BW * bin_size * bin_size;
  const int num_threads = gridDim.x * blockDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;

  for (int pid = tid; pid < num_pixels; pid += num_threads) {
    // Convert linear index into bin and pixel indices. We make the within
    // block pixel ids move the fastest, so that adjacent threads will fall
    // into the same bin; this should give them coalesced memory reads when
    // they read from points and bin_points.
    int i = pid;
    const int n = i / (BH * BW * bin_size * bin_size);
    i %= BH * BW * bin_size * bin_size;
    const int by = i / (BW * bin_size * bin_size);
    i %= BW * bin_size * bin_size;
    const int bx = i / (bin_size * bin_size);
    i %= bin_size * bin_size;

    const int yi = i / bin_size + by * bin_size;
    const int xi = i % bin_size + bx * bin_size;

    if (yi >= H || xi >= W)
      continue;

    const float xf = PixToNonSquareNdc(xi, W, H);
    const float yf = PixToNonSquareNdc(yi, H, W);

    // This part looks like the naive rasterization kernel, except we use
    // bin_points to only look at a subset of points already known to fall
    // in this bin. TODO abstract out this logic into some data structure
    // that is shared by both kernels?
    Pix q[kMaxPointsPerPixel];
    int q_size = 0;
    float q_max_z = -1000;
    int q_max_idx = -1;
    for (int m = 0; m < M; ++m) {
      const int p = bin_points[n * BH * BW * M + by * BW * M + bx * M + m];
      if (p < 0) {
        // bin_points uses -1 as a sentinal value
        continue;
      }
      CheckPixelInsidePoint(
          points, p, q_size, q_max_z, q_max_idx, q, radius, xf, yf, K);
    }
    // Now we've looked at all the points for this bin, so we can write
    // output for the current pixel.
    BubbleSort(q, q_size);

    // Reverse ordering of the X and Y axis as the camera coordinates
    // assume that +Y is pointing up and +X is pointing left.
    const int yidx = H - 1 - yi;
    const int xidx = W - 1 - xi;

    const int pix_idx = n * H * W * K + yidx * W * K + xidx * K;
    for (int k = 0; k < q_size; ++k) {
      point_idxs[pix_idx + k] = q[k].idx;
      zbuf[pix_idx + k] = q[k].z;
      pix_dists[pix_idx + k] = q[k].dist2;
    }
  }
}

std::tuple<at::Tensor, at::Tensor, at::Tensor> RasterizePointsFineCuda(
    const at::Tensor& points, // (P, 3)
    const at::Tensor& bin_points,
    const std::tuple<int, int> image_size,
    const at::Tensor& radius,
    const int bin_size,
    const int points_per_pixel) {
  // Check inputs are on the same device
  at::TensorArg points_t{points, "points", 1},
      bin_points_t{bin_points, "bin_points", 2};
  at::CheckedFrom c = "RasterizePointsFineCuda";
  at::checkAllSameGPU(c, {points_t, bin_points_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(points.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int N = bin_points.size(0);
  const int BH = bin_points.size(1);
  const int BW = bin_points.size(2);
  const int M = bin_points.size(3);
  const int K = points_per_pixel;

  const int H = std::get<0>(image_size);
  const int W = std::get<1>(image_size);

  if (K > kMaxPointsPerPixel) {
    AT_ERROR("Must have num_closest <= 150");
  }
  auto int_opts = bin_points.options().dtype(at::kInt);
  auto float_opts = points.options().dtype(at::kFloat);
  at::Tensor point_idxs = at::full({N, H, W, K}, -1, int_opts);
  at::Tensor zbuf = at::full({N, H, W, K}, -1, float_opts);
  at::Tensor pix_dists = at::full({N, H, W, K}, -1, float_opts);

  if (point_idxs.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(point_idxs, zbuf, pix_dists);
  }

  const size_t blocks = 1024;
  const size_t threads = 64;
  RasterizePointsFineCudaKernel<<<blocks, threads, 0, stream>>>(
      points.contiguous().data_ptr<float>(),
      bin_points.contiguous().data_ptr<int32_t>(),
      radius.contiguous().data_ptr<float>(),
      bin_size,
      N,
      BH,
      BW,
      M,
      H,
      W,
      K,
      point_idxs.contiguous().data_ptr<int32_t>(),
      zbuf.contiguous().data_ptr<float>(),
      pix_dists.contiguous().data_ptr<float>());

  AT_CUDA_CHECK(hipGetLastError());
  return std::make_tuple(point_idxs, zbuf, pix_dists);
}

// ****************************************************************************
// *                            BACKWARD PASS                                 *
// ****************************************************************************
// TODO(T55115174) Add more documentation for backward kernel.
__global__ void RasterizePointsBackwardCudaKernel(
    const float* points, // (P, 3)
    const int32_t* idxs, // (N, H, W, K)
    const int N,
    const int P,
    const int H,
    const int W,
    const int K,
    const float* grad_zbuf, // (N, H, W, K)
    const float* grad_dists, // (N, H, W, K)
    float* grad_points) { // (P, 3)
  // Parallelized over each of K points per pixel, for each pixel in images of
  // size H * W, for each image in the batch of size N.
  int num_threads = gridDim.x * blockDim.x;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = tid; i < N * H * W * K; i += num_threads) {
    // const int n = i / (H * W * K); // batch index (not needed).
    const int yxk = i % (H * W * K);
    const int yi = yxk / (W * K);
    const int xk = yxk % (W * K);
    const int xi = xk / K;
    // k = xk % K (We don't actually need k, but this would be it.)
    // Reverse ordering of X and Y axes.
    const int yidx = H - 1 - yi;
    const int xidx = W - 1 - xi;

    const float xf = PixToNonSquareNdc(xidx, W, H);
    const float yf = PixToNonSquareNdc(yidx, H, W);

    const int p = idxs[i];
    if (p < 0)
      continue;
    const float grad_dist2 = grad_dists[i];
    const int p_ind = p * 3; // index into packed points tensor
    const float px = points[p_ind + 0];
    const float py = points[p_ind + 1];
    const float dx = px - xf;
    const float dy = py - yf;
    const float grad_px = 2.0f * grad_dist2 * dx;
    const float grad_py = 2.0f * grad_dist2 * dy;
    const float grad_pz = grad_zbuf[i];
    atomicAdd(grad_points + p_ind + 0, grad_px);
    atomicAdd(grad_points + p_ind + 1, grad_py);
    atomicAdd(grad_points + p_ind + 2, grad_pz);
  }
}

at::Tensor RasterizePointsBackwardCuda(
    const at::Tensor& points, // (N, P, 3)
    const at::Tensor& idxs, // (N, H, W, K)
    const at::Tensor& grad_zbuf, // (N, H, W, K)
    const at::Tensor& grad_dists) { // (N, H, W, K)

  // Check inputs are on the same device
  at::TensorArg points_t{points, "points", 1}, idxs_t{idxs, "idxs", 2},
      grad_zbuf_t{grad_zbuf, "grad_zbuf", 3},
      grad_dists_t{grad_dists, "grad_dists", 4};
  at::CheckedFrom c = "RasterizePointsBackwardCuda";
  at::checkAllSameGPU(c, {points_t, idxs_t, grad_zbuf_t, grad_dists_t});
  at::checkAllSameType(c, {points_t, grad_zbuf_t, grad_dists_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(points.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int P = points.size(0);
  const int N = idxs.size(0);
  const int H = idxs.size(1);
  const int W = idxs.size(2);
  const int K = idxs.size(3);

  at::Tensor grad_points = at::zeros({P, 3}, points.options());

  if (grad_points.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return grad_points;
  }

  const size_t blocks = 1024;
  const size_t threads = 64;

  RasterizePointsBackwardCudaKernel<<<blocks, threads, 0, stream>>>(
      points.contiguous().data_ptr<float>(),
      idxs.contiguous().data_ptr<int32_t>(),
      N,
      P,
      H,
      W,
      K,
      grad_zbuf.contiguous().data_ptr<float>(),
      grad_dists.contiguous().data_ptr<float>(),
      grad_points.contiguous().data_ptr<float>());

  AT_CUDA_CHECK(hipGetLastError());
  return grad_points;
}
